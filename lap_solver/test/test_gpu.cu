#include "hip/hip_runtime.h"
#define LAP_CUDA
#define LAP_QUIET
//#define LAP_DISPLAY_EVALUATED
//#define LAP_DEBUG
//#define LAP_NO_MEM_DEBUG
//#define LAP_ROWS_SCANNED
// these two don't work together at the moment
#ifndef LAP_ROWS_SCANNED
# define LAP_CUDA_LOCAL_ROWSOL
#endif
// should only be enabled for testing purposes
//#define LAP_CUDA_ALLOW_WDDM
// enable one thread per GPU
#define LAP_CUDA_OPENMP

#include "../lap.h"

#include <random>
#include <string>
#include "test_options.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <class C> void testSanityCached(long long min_cached, long long max_cached, long long max_memory, int runs, bool epsilon, std::string name_C, std::vector<int> &devs, bool silent);
template <class C> void testGeometricCached(long long min_cached, long long max_cached, long long max_memory, int runs, bool epsilon, bool disjoint, std::string name_C, std::vector<int> &devs, bool silent);
template <class C> void testRandomLowRankCached(long long min_cached, long long max_cached, long long max_memory, long long min_rank, long long max_rank, int runs, bool epsilon, std::string name_C, std::vector<int> &devs, bool silent);
template <class C> void testInteger(long long min_tab, long long max_tab, long long max_memory, int runs, bool epsilon, std::string name_C, std::vector<int> &devs, bool silent);

int main(int argc, char* argv[])
{
	Options opt;
	int r = opt.parseOptions(argc, argv);
	if (r != 0) return r;

	if (opt.use_double)
	{
		if (opt.use_single)
		{
			//if (opt.run_sanity) testSanity<double>(opt.lap_min_tab, opt.lap_max_tab, opt.runs, false, std::string("double"));
			if (opt.run_sanity_cached) testSanityCached<double>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.runs, false, std::string("double"), opt.devices, opt.silent);
			//if (opt.run_random) testRandom<double>(opt.lap_min_tab, opt.lap_max_tab, opt.runs, false, std::string("double"));
			//if (opt.run_random_low_rank) testRandomLowRank<double>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_min_rank, opt.lap_max_rank, opt.runs, false, std::string("double"));
			if (opt.run_random_low_rank_cached) testRandomLowRankCached<double>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.lap_min_rank, opt.lap_max_rank, opt.runs, false, std::string("double"), opt.devices, opt.silent);
			//if (opt.run_geometric) testGeometric<double>(opt.lap_min_tab, opt.lap_max_tab, opt.runs, false, false, std::string("double"));
			//if (opt.run_geometric_disjoint) testGeometric<double>(opt.lap_min_tab, opt.lap_max_tab, opt.runs, false, true, std::string("double"));
			if (opt.run_geometric_cached) testGeometricCached<double>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.runs, false, false, std::string("double"), opt.devices, opt.silent);
			if (opt.run_geometric_disjoint_cached) testGeometricCached<double>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.runs, false, true, std::string("double"), opt.devices, opt.silent);
			//if (opt.images.size() > 1) testImages<double>(opt.images, opt.lap_max_memory, opt.runs, false, std::string("double"));
		}
		if (opt.use_epsilon)
		{
			//if (opt.run_sanity) testSanity<double>(opt.lap_min_tab, opt.lap_max_tab, opt.runs, true, std::string("double"));
			if (opt.run_sanity_cached) testSanityCached<double>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.runs, true, std::string("double"), opt.devices, opt.silent);
			//if (opt.run_random) testRandom<double>(opt.lap_min_tab, opt.lap_max_tab, opt.runs, true, std::string("double"));
			//if (opt.run_random_low_rank) testRandomLowRank<double>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_min_rank, opt.lap_max_rank, opt.runs, true, std::string("double"));
			if (opt.run_random_low_rank_cached) testRandomLowRankCached<double>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.lap_min_rank, opt.lap_max_rank, opt.runs, true, std::string("double"), opt.devices, opt.silent);
			//if (opt.run_geometric) testGeometric<double>(opt.lap_min_tab, opt.lap_max_tab, opt.runs, true, false, std::string("double"));
			//if (opt.run_geometric_disjoint) testGeometric<double>(opt.lap_min_tab, opt.lap_max_tab, opt.runs, true, true, std::string("double"));
			if (opt.run_geometric_cached) testGeometricCached<double>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.runs, true, false, std::string("double"), opt.devices, opt.silent);
			if (opt.run_geometric_disjoint_cached) testGeometricCached<double>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.runs, true, true, std::string("double"), opt.devices, opt.silent);
			//if (opt.images.size() > 1) testImages<double>(opt.images, opt.lap_max_memory, opt.runs, true, std::string("double"));
		}
	}
	if (opt.use_float)
	{
		if (opt.use_single)
		{
			//if (opt.run_sanity) testSanity<float>(opt.lap_min_tab, opt.lap_max_tab, opt.runs, false, std::string("float"));
			if (opt.run_sanity_cached) testSanityCached<float>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.runs, false, std::string("float"), opt.devices, opt.silent);
			//if (opt.run_random) testRandom<float>(opt.lap_min_tab, opt.lap_max_tab, opt.runs, false, std::string("float"));
			//if (opt.run_random_low_rank) testRandomLowRank<float>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_min_rank, opt.lap_max_rank, opt.runs, false, std::string("float"));
			if (opt.run_random_low_rank_cached) testRandomLowRankCached<float>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.lap_min_rank, opt.lap_max_rank, opt.runs, false, std::string("float"), opt.devices, opt.silent);
			//if (opt.run_geometric) testGeometric<float>(opt.lap_min_tab, opt.lap_max_tab, opt.runs, false, false, std::string("float"));
			//if (opt.run_geometric_disjoint) testGeometric<float>(opt.lap_min_tab, opt.lap_max_tab, opt.runs, false, true, std::string("float"));
			if (opt.run_geometric_cached) testGeometricCached<float>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.runs, false, false, std::string("float"), opt.devices, opt.silent);
			if (opt.run_geometric_disjoint_cached) testGeometricCached<float>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.runs, false, true, std::string("float"), opt.devices, opt.silent);
			//if (opt.images.size() > 1) testImages<float>(opt.images, opt.lap_max_memory, opt.runs, false, std::string("float"));
		}
		if (opt.use_epsilon)
		{
			//if (opt.run_sanity) testSanity<float>(opt.lap_min_tab, opt.lap_max_tab, opt.runs, true, std::string("float"));
			if (opt.run_sanity_cached) testSanityCached<float>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.runs, true, std::string("float"), opt.devices, opt.silent);
			//if (opt.run_random) testRandom<float>(opt.lap_min_tab, opt.lap_max_tab, opt.runs, true, std::string("float"));
			//if (opt.run_random_low_rank) testRandomLowRank<float>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_min_rank, opt.lap_max_rank, opt.runs, true, std::string("float"));
			if (opt.run_random_low_rank_cached) testRandomLowRankCached<float>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.lap_min_rank, opt.lap_max_rank, opt.runs, true, std::string("float"), opt.devices, opt.silent);
			//if (opt.run_geometric) testGeometric<float>(opt.lap_min_tab, opt.lap_max_tab, opt.runs, true, false, std::string("float"));
			//if (opt.run_geometric_disjoint) testGeometric<float>(opt.lap_min_tab, opt.lap_max_tab, opt.runs, true, true, std::string("float"));
			if (opt.run_geometric_cached) testGeometricCached<float>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.runs, true, false, std::string("float"), opt.devices, opt.silent);
			if (opt.run_geometric_disjoint_cached) testGeometricCached<float>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.runs, true, true, std::string("float"), opt.devices, opt.silent);
			//if (opt.images.size() > 1) testImages<float>(opt.images, opt.lap_max_memory, opt.runs, true, std::string("float"));
		}
	}
	if (opt.run_integer)
	{
		if (opt.use_double)
		{
			if (opt.use_single) testInteger<double>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, false, std::string("double"), opt.devices, opt.silent);
			if (opt.use_epsilon) testInteger<double>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, true, std::string("double"), opt.devices, opt.silent);
		}
		if (opt.use_float)
		{
			if (opt.use_single) testInteger<float>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, false, std::string("float"), opt.devices, opt.silent);
			if (opt.use_epsilon) testInteger<float>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, true, std::string("float"), opt.devices, opt.silent);
		}
		if (opt.use_single) testInteger<long long>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, false, std::string("long long"), opt.devices, opt.silent);
		if (opt.use_epsilon) testInteger<long long>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, true, std::string("long long"), opt.devices, opt.silent);
	}

	return 0;
}

template <class C>
__global__
void getCostRow_geometric_kernel(C *cost, C *tab_s, C *tab_t, int x, int start, int end, int N)
{
	int y = start + threadIdx.x + blockIdx.x * blockDim.x;
	if (y >= end) return;

	C d0 = tab_s[x] - tab_t[y];
	C d1 = tab_s[x + N] - tab_t[y + N];
	cost[threadIdx.x + blockIdx.x * blockDim.x] = d0 * d0 + d1 * d1;
}

template <class C>
__global__
void getCost_geometric_kernel(C *cost, C *tab_s, C *tab_t, int *rowsol, int N)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	if (x >= N) return;
	int y = rowsol[x];

	float d0 = tab_s[x] - tab_t[y];
	float d1 = tab_s[x + N] - tab_t[y + N];
	cost[x] = d0 * d0 + d1 * d1;
}

template <class C>
void testGeometricCached(long long min_cached, long long max_cached, long long max_memory, int runs, bool epsilon, bool disjoint, std::string name_C, std::vector<int> &devs, bool silent)
{
	for (long long NN = min_cached * min_cached; NN <= max_cached * max_cached; NN <<= 1)
	{
		for (int r = 0; r < runs; r++)
		{
			int N = (int)floor(sqrt((double)NN));

			std::cout << "Geometric";
			if (disjoint) std::cout << " Disjoint";
			std::cout << " R^2<" << name_C << "> " << N << "x" << N << " (" << (double)max_memory / 1073741824.0 << "GB / GPU)";
			if (epsilon) std::cout << " with epsilon scaling";
			std::cout << std::endl;

			auto start_time = std::chrono::high_resolution_clock::now();

			std::uniform_real_distribution<C> distribution(0.0, 1.0);
			std::mt19937_64 generator(1234);

			C *tab_s = new C[2 * N];
			C *tab_t = new C[2 * N];
			for (int i = 0; i < N; i++)
			{
				tab_s[i] = distribution(generator);
				tab_t[i] = distribution(generator);
				tab_s[i + N] = distribution(generator);
				tab_t[i + N] = distribution(generator);
			}

			// order of coordinates is different, first all x then all y
			if (disjoint)
			{
				for (int i = 0; i < N; i++)
				{
					if ((i << 1) < N)
					{
						tab_t[i] += C(1.0);
					}
					else
					{
						tab_s[i] += C(1.0);
						tab_s[i + N] += C(1.0);
						tab_t[i + N] += C(1.0);
					}
				}
			}

			// enabled function
			lap::cuda::Worksharing ws(N, 256, devs, silent);
			int num_enabled = (int)ws.device.size();

			int step = (int)N / (int)std::min((long long)N, (long long)((num_enabled * max_memory) / (sizeof(C) * N)));

			C **d_tab_s = new C*[num_enabled];
			C **d_tab_t = new C*[num_enabled];

			for (int i = 0; i < num_enabled; i++)
			{
				d_tab_s[i] = 0;
				d_tab_t[i] = 0;
			}

			for (int i = 0; i < num_enabled; i++)
			{
				hipSetDevice(ws.device[i]);
				hipMalloc(&(d_tab_s[i]), 2 * N * sizeof(C));
				hipMalloc(&(d_tab_t[i]), 2 * N * sizeof(C));
				hipMemcpy(d_tab_s[i], tab_s, 2 * N * sizeof(C), hipMemcpyHostToDevice);
				hipMemcpy(d_tab_t[i], tab_t, 2 * N * sizeof(C), hipMemcpyHostToDevice);
			}

			int *rowsol = new int[N];

			// cost function
			auto get_cost_row = [&d_tab_s, &d_tab_t, &N](C *d_row, int t, hipStream_t stream, int x, int start, int end)
			{
				dim3 block_size, grid_size;
				block_size.x = 256;
				grid_size.x = ((end - start) + block_size.x - 1) / block_size.x;
				getCostRow_geometric_kernel<<<grid_size, block_size, 0, stream>>>(d_row, d_tab_s[t], d_tab_t[t], x, start, end, N);
			};

			lap::cuda::RowCostFunction<C, decltype(get_cost_row)> costFunction(get_cost_row);

			// different cache size, so always use SLRU
			lap::cuda::CachingIterator<C, C, decltype(costFunction), lap::CacheSLRU> iterator(N, N, max_memory / sizeof(C), costFunction, ws);
			lap::displayTime(start_time, "setup complete", std::cout);
			if (epsilon) costFunction.setInitialEpsilon(lap::cuda::guessEpsilon<C, C>(N, N, iterator));

			lap::cuda::solve<C, C>(N, costFunction, iterator, rowsol);

			{
				// set device back to 0
				hipSetDevice(ws.device[0]);
				std::stringstream ss;
				C my_cost(0);
				C *row = new C[N];
				// calculate costs directly
				{
					int *d_rowsol;
					C *d_row;
					hipMalloc(&d_rowsol, N * sizeof(int));
					hipMalloc(&d_row, N * sizeof(C));
					hipMemcpy(d_rowsol, rowsol, N * sizeof(int), hipMemcpyHostToDevice);
					dim3 block_size, grid_size;
					block_size.x = 256;
					grid_size.x = (N + block_size.x - 1) / block_size.x;
					getCost_geometric_kernel<<<grid_size, block_size>>>(d_row, d_tab_s[0], d_tab_t[0], d_rowsol, N);
					hipMemcpy(row, d_row, N * sizeof(C), hipMemcpyDeviceToHost);
					hipFree(d_row);
					hipFree(d_rowsol);
				}
				for (int i = 0; i < N; i++) my_cost += row[i];
				delete[] row;
				ss << "cost = " << my_cost;// lap::cost<C, float>(N, costFunction, rowsol);
				lap::displayTime(start_time, ss.str().c_str(), std::cout);
			}

			for (int i = 0; i < num_enabled; i++)
			{
				hipSetDevice(ws.device[i]);
				hipFree(d_tab_s[i]);
				hipFree(d_tab_t[i]);
			}

			delete[] rowsol;
			delete[] tab_s;
			delete[] tab_t;
			delete[] d_tab_s;
			delete[] d_tab_t;
		}
	}
}

template <class C>
__global__
void getCostRow_sanity_kernel(C *cost, C *vec, int x, int start, int end, int N)
{
	int y = start + threadIdx.x + blockIdx.x * blockDim.x;
	if (y >= end) return;

	C r = vec[x] + vec[y + N];
	if (x != y) r += C(0.1);

	cost[threadIdx.x + blockIdx.x * blockDim.x] = r;
}

template <class C>
__global__
void getCost_sanity_kernel(C *cost, C *vec, int *rowsol, int N)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	if (x >= N) return;
	int y = rowsol[x];

	C r = vec[x] + vec[y + N];
	if (x != y) r += C(0.1);

	cost[x] = r;
}

template <class C>
__global__
void getGTCost_sanity_kernel(C *cost, C *vec, int N)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	if (x >= N) return;
	int y = x;

	C r = vec[x] + vec[y + N];
	if (x != y) r += C(0.1);

	cost[x] = r;
}

template <class C>
void testSanityCached(long long min_cached, long long max_cached, long long max_memory, int runs, bool epsilon, std::string name_C, std::vector<int> &devs, bool silent)
{
	for (long long NN = min_cached * min_cached; NN <= max_cached * max_cached; NN <<= 1)
	{
		for (int r = 0; r < runs; r++)
		{
			int N = (int)floor(sqrt((double)NN));

			std::cout << "Sanity<" << name_C << "> " << N << "x" << N << " (" << (double)max_memory / 1073741824.0 << "GB / GPU)";
			if (epsilon) std::cout << " with epsilon scaling";
			std::cout << std::endl;

			auto start_time = std::chrono::high_resolution_clock::now();

			std::uniform_real_distribution<C> distribution(0.0, 1.0);
			std::mt19937_64 generator(1234);

			C *vec = new C[N << 1];

			for (long long i = 0; i < N << 1; i++) vec[i] = distribution(generator);

			// enabled function
			lap::cuda::Worksharing ws(N, 256, devs, silent);
			int num_enabled = (int)ws.device.size();

			int step = (int)N / (int)std::min((long long)N, (long long)((num_enabled * max_memory) / (sizeof(C) * N)));

			C **d_vec = new C*[num_enabled];

			for (int i = 0; i < num_enabled; i++)
			{
				d_vec[i] = 0;
			}

			for (int i = 0; i < num_enabled; i++)
			{
				hipSetDevice(ws.device[i]);
				hipMalloc(&(d_vec[i]), 2 * N * sizeof(C));
				hipMemcpy(d_vec[i], vec, 2 * N * sizeof(C), hipMemcpyHostToDevice);
			}

			int *rowsol = new int[N];

			// cost function
			auto get_cost_row = [&d_vec, &N](C *d_row, int t, hipStream_t stream, int x, int start, int end)
			{
				dim3 block_size, grid_size;
				block_size.x = 256;
				grid_size.x = ((end - start) + block_size.x - 1) / block_size.x;
				getCostRow_sanity_kernel<<<grid_size, block_size, 0, stream>>>(d_row, d_vec[t], x, start, end, N);
			};

			lap::cuda::RowCostFunction<C, decltype(get_cost_row)> costFunction(get_cost_row);

			// different cache size, so always use SLRU
			lap::cuda::CachingIterator<C, C, decltype(costFunction), lap::CacheSLRU> iterator(N, N, max_memory / sizeof(C), costFunction, ws);
			lap::displayTime(start_time, "setup complete", std::cout);
			if (epsilon) costFunction.setInitialEpsilon(lap::cuda::guessEpsilon<C, C>(N, N, iterator));

			lap::cuda::solve<C, C>(N, costFunction, iterator, rowsol);

			{
				// set device back to 0
				hipSetDevice(ws.device[0]);
				std::stringstream ss;
				C my_cost(0);
				C *row = new C[N];
				// calculate costs directly
				{
					int *d_rowsol;
					C *d_row;
					hipMalloc(&d_rowsol, N * sizeof(int));
					hipMalloc(&d_row, N * sizeof(C));
					hipMemcpy(d_rowsol, rowsol, N * sizeof(int), hipMemcpyHostToDevice);
					dim3 block_size, grid_size;
					block_size.x = 256;
					grid_size.x = (N + block_size.x - 1) / block_size.x;
					getCost_sanity_kernel<<<grid_size, block_size>>>(d_row, d_vec[0], d_rowsol, N);
					hipMemcpy(row, d_row, N * sizeof(C), hipMemcpyDeviceToHost);
					hipFree(d_row);
					hipFree(d_rowsol);
				}
				for (int i = 0; i < N; i++) my_cost += row[i];
				delete[] row;
				ss << "cost = " << my_cost;
				lap::displayTime(start_time, ss.str().c_str(), std::cout);
			}

			bool passed = true;
			for (long long i = 0; (passed) && (i < N); i++)
			{
				passed &= (rowsol[i] == i);
			}
			std::stringstream ss;
			if (passed) ss << "test passed: ";
			else ss << "test failed: ";
			{
				// set device back to 0
				hipSetDevice(ws.device[0]);
				C my_cost(0);
				C *row = new C[N];
				// calculate costs directly
				{
					C *d_row;
					hipMalloc(&d_row, N * sizeof(C));
					dim3 block_size, grid_size;
					block_size.x = 256;
					grid_size.x = (N + block_size.x - 1) / block_size.x;
					getGTCost_sanity_kernel<<<grid_size, block_size>>>(d_row, d_vec[0], N);
					hipMemcpy(row, d_row, N * sizeof(C), hipMemcpyDeviceToHost);
					hipFree(d_row);
				}
				for (int i = 0; i < N; i++) my_cost += row[i];
				delete[] row;
				ss << "ground truth cost = " << my_cost;
			}
			lap::displayTime(start_time, ss.str().c_str(), std::cout);

			for (int i = 0; i < num_enabled; i++)
			{
				hipSetDevice(ws.device[i]);
				hipFree(d_vec[i]);
			}

			delete[] rowsol;
			delete[] vec;
			delete[] d_vec;
		}
	}
}

template <class C>
__global__
void getCostRow_lowRank_kernel(C *cost, C *vec, int rank, int x, int start, int end, int N)
{
	int y = start + threadIdx.x + blockIdx.x * blockDim.x;
	if (y >= end) return;

	C sum(0);
	for (long long k = 0; k < rank; k++)
	{
		sum += vec[k * N + x] * vec[k * N + y];
	}
	sum /= C(rank);

	cost[threadIdx.x + blockIdx.x * blockDim.x] = sum;
}

template <class C>
__global__
void getCost_lowRank_kernel(C *cost, C *vec, int rank, int *rowsol, int N)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	if (x >= N) return;
	int y = rowsol[x];

	C sum(0);
	for (long long k = 0; k < rank; k++)
	{
		sum += vec[k * N + x] * vec[k * N + y];
	}
	sum /= C(rank);

	cost[x] = sum;
}

template <class C>
void testRandomLowRankCached(long long min_cached, long long max_cached, long long max_memory, long long min_rank, long long max_rank, int runs, bool epsilon, std::string name_C, std::vector<int> &devs, bool silent)
{
	for (long long rank = min_rank; rank <= max_rank; rank <<= 1)
	{
		for (long long NN = min_cached * min_cached; NN <= max_cached * max_cached; NN <<= 1)
		{
			for (int r = 0; r < runs; r++)
			{
				int N = (int)floor(sqrt((double)NN));
				int entries = (int)std::min((long long)N, (long long)(max_memory / (sizeof(C) * N)));

				std::cout << "RandomLowRank<" << name_C << "> " << N << "x" << N << " (" << entries << ") rank = " << rank;
				if (epsilon) std::cout << " with epsilon scaling";
				std::cout << std::endl;

				auto start_time = std::chrono::high_resolution_clock::now();

				std::uniform_real_distribution<C> distribution(0.0, 1.0);
				std::mt19937_64 generator(1234);

				// The following matrix will have at most the seletcted rank.
				C *vec = new C[N * rank];
				for (long long i = 0; i < rank; i++)
				{
					for (long long j = 0; j < N; j++) vec[i * N + j] = distribution(generator);
				}

				// enabled function
				lap::cuda::Worksharing ws(N, 256, devs, silent);
				int num_enabled = (int)ws.device.size();

				int step = (int)N / (int)std::min((long long)N, (long long)((num_enabled * max_memory) / (sizeof(C) * N)));

				C **d_vec = new C*[num_enabled];

				for (int i = 0; i < num_enabled; i++)
				{
					d_vec[i] = 0;
				}

				for (int i = 0; i < num_enabled; i++)
				{
					hipSetDevice(ws.device[i]);
					hipMalloc(&(d_vec[i]), 2 * N * sizeof(C));
					hipMemcpy(d_vec[i], vec, 2 * N * sizeof(C), hipMemcpyHostToDevice);
				}

				int *rowsol = new int[N];

				// cost function
				auto get_cost_row = [&d_vec, &N, &rank](C *d_row, int t, hipStream_t stream, int x, int start, int end)
				{
					dim3 block_size, grid_size;
					block_size.x = 256;
					grid_size.x = ((end - start) + block_size.x - 1) / block_size.x;
					getCostRow_lowRank_kernel<<<grid_size, block_size, 0, stream>>>(d_row, d_vec[t], (int)rank, x, start, end, N);
				};

				lap::cuda::RowCostFunction<C, decltype(get_cost_row)> costFunction(get_cost_row);

				// different cache size, so always use SLRU
				lap::cuda::CachingIterator<C, C, decltype(costFunction), lap::CacheSLRU> iterator(N, N, max_memory / sizeof(C), costFunction, ws);
				lap::displayTime(start_time, "setup complete", std::cout);
				if (epsilon) costFunction.setInitialEpsilon(lap::cuda::guessEpsilon<C, C>(N, N, iterator));

				lap::cuda::solve<C, C>(N, costFunction, iterator, rowsol);

				{
					// set device back to 0
					hipSetDevice(ws.device[0]);
					std::stringstream ss;
					C my_cost(0);
					C *row = new C[N];
					// calculate costs directly
					{
						int *d_rowsol;
						C *d_row;
						hipMalloc(&d_rowsol, N * sizeof(int));
						hipMalloc(&d_row, N * sizeof(C));
						hipMemcpy(d_rowsol, rowsol, N * sizeof(int), hipMemcpyHostToDevice);
						dim3 block_size, grid_size;
						block_size.x = 256;
						grid_size.x = (N + block_size.x - 1) / block_size.x;
						getCost_lowRank_kernel<<<grid_size, block_size>>>(d_row, d_vec[0], (int)rank, d_rowsol, N);
						hipMemcpy(row, d_row, N * sizeof(C), hipMemcpyDeviceToHost);
						hipFree(d_row);
						hipFree(d_rowsol);
					}
					for (int i = 0; i < N; i++) my_cost += row[i];
					delete[] row;
					ss << "cost = " << my_cost;// lap::cost<C, float>(N, costFunction, rowsol);
					lap::displayTime(start_time, ss.str().c_str(), std::cout);
				}

				for (int i = 0; i < num_enabled; i++)
				{
					hipSetDevice(ws.device[i]);
					hipFree(d_vec[i]);
				}

				delete[] rowsol;
				delete[] vec;
				delete[] d_vec;
			}
		}
	}
}

template <class C>
void testInteger(long long min_tab, long long max_tab, long long max_memory, int runs, bool epsilon, std::string name_C, std::vector<int> &devs, bool silent)
{
	// random costs (directly supply cost matrix)
	for (int range = 0; range < 3; range++)
	{
		for (long long NN = min_tab * min_tab; NN <= max_tab * max_tab; NN <<= 1)
		{
			for (int r = 0; r < runs; r++)
			{
				int N = (int)floor(sqrt((double)NN));

				std::cout << "Integer";
				std::cout << "<" << name_C << " ";
				if (range == 0) std::cout << "1/10n";
				else if (range == 1) std::cout << "n";
				else std::cout << "10n";
				std::cout << "> " << N << "x" << N << " table";
				if (epsilon) std::cout << " with epsilon scaling";
				std::cout << std::endl;

				int n;
				if (range == 0) n = N / 10;
				else if (range == 1) n = N;
				else n = 10 * N;
				std::uniform_int_distribution<int> distribution(0, n);
				std::mt19937_64 generator(1234);

				auto start_time = std::chrono::high_resolution_clock::now();

				auto get_cost = [&distribution, &generator](int x, int y) -> int
				{
					return distribution(generator);
				};

				lap::cuda::Worksharing ws(N, 256, devs, silent);
				int num_enabled = (int)ws.device.size();

				int step = (int)N / (int)std::min((long long)N, (long long)((num_enabled * max_memory) / (sizeof(C) * N)));

				int *rowsol = new int[N];

				lap::SimpleCostFunction<int, decltype(get_cost)> cpuCostFunction(get_cost);
				lap::TableCost<int> costMatrix(N, N, cpuCostFunction);

				// cost function (copy data from table)
				auto get_cost_row = [&costMatrix](int *d_row, int t, hipStream_t stream, int x, int start, int end)
				{
					hipMemcpyAsync(d_row, costMatrix.getRow(x) + start, (end - start) * sizeof(int), hipMemcpyHostToDevice, stream);
				};

				lap::cuda::RowCostFunction<int, decltype(get_cost_row)> costFunction(get_cost_row);

				// different cache size, so always use SLRU
				lap::cuda::CachingIterator<C, int, decltype(costFunction), lap::CacheSLRU> iterator(N, N, max_memory / sizeof(int), costFunction, ws);
				lap::displayTime(start_time, "setup complete", std::cout);
				if (epsilon) costFunction.setInitialEpsilon((int)lap::cuda::guessEpsilon<C, int>(N, N, iterator));

				lap::cuda::solve<C, int>(N, costFunction, iterator, rowsol);

				std::stringstream ss;
				ss << "cost = " << lap::cost<C>(N, N, costMatrix, rowsol);
				lap::displayTime(start_time, ss.str().c_str(), std::cout);

				delete[] rowsol;
			}
		}
	}
}
